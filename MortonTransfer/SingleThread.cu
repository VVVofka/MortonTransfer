#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>
#include <iostream>
#include <vector>
#include <random>
#include <cassert>
#include <chrono>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

static __device__ __forceinline__ unsigned DecodeMorton2X(unsigned code){
    code &= 0x55555555;
    code = (code ^ (code >> 1)) & 0x33333333;
    code = (code ^ (code >> 2)) & 0x0F0F0F0F;
    code = (code ^ (code >> 4)) & 0x00FF00FF;
    code = (code ^ (code >> 8)) & 0x0000FFFF;
    return code;
}

static __device__ __forceinline__ unsigned DecodeMorton2Y(unsigned code){
    code >>= 1;
    code &= 0x55555555;
    code = (code ^ (code >> 1)) & 0x33333333;
    code = (code ^ (code >> 2)) & 0x0F0F0F0F;
    code = (code ^ (code >> 4)) & 0x00FF00FF;
    code = (code ^ (code >> 8)) & 0x0000FFFF;
    return code;
}

static __device__ __forceinline__ unsigned EncodeMorton2(unsigned x, unsigned y){
    x &= 0x0000ffff;
    y &= 0x0000ffff;
    x = (x | (x << 8)) & 0x00FF00FF;
    y = (y | (y << 8)) & 0x00FF00FF;
    x = (x | (x << 4)) & 0x0F0F0F0F;
    y = (y | (y << 4)) & 0x0F0F0F0F;
    x = (x | (x << 2)) & 0x33333333;
    y = (y | (y << 2)) & 0x33333333;
    x = (x | (x << 1)) & 0x55555555;
    y = (y | (y << 1)) & 0x55555555;
    return x | (y << 1);
}

//static __device__ __forceinline__ unsigned get2bits64(const uint64_t* data, unsigned index){
//    unsigned word_index = index >> 5;
//    unsigned bit_offset = (index & 31) << 1;
//    return (data[word_index] >> bit_offset) & 0x3;
//}

static __device__ __forceinline__ void set2bits64(uint64_t* data, unsigned index, unsigned value){
    unsigned word_index = index >> 5;
    unsigned bit_offset = (index & 31) << 1;
    uint64_t mask = ~(0x3ULL << bit_offset);
    data[word_index] = (data[word_index] & mask) | ((uint64_t)(value & 0x3) << bit_offset);
}

__global__ void transfer64_tile(const uint64_t* __restrict__ data_in,
                                uint64_t* __restrict__ data_out,
                                int2 shift,
                                unsigned size_side){
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned total_values = size_side * size_side;
    unsigned total_words = (total_values + 31) / 32;

    if(tid >= total_words) return;

    uint64_t word = data_in[tid];

    // 32 значения по 2 бита
#pragma unroll
    for(unsigned i = 0; i < 32; ++i){
        unsigned global_index = (tid << 5) + i;
        if(global_index >= total_values) break;

        unsigned val = (word >> (i * 2)) & 0x3;

        unsigned x = DecodeMorton2X(global_index);
        unsigned y = DecodeMorton2Y(global_index);

        x = (x + shift.x + size_side) % size_side;
        y = (y + shift.y + size_side) % size_side;

        unsigned new_index = EncodeMorton2(x, y);
        set2bits64(data_out, new_index, val);
    }
}
static unsigned get2bits_host(const std::vector<uint64_t>& buffer, unsigned index){
	unsigned word_index = index / 32;
	unsigned bit_offset = (index % 32) * 2;
	return (buffer[word_index] >> bit_offset) & 0x3;
}
static void set2bits_host(std::vector<uint64_t>& buffer, unsigned index, unsigned value){
	unsigned word_index = index / 32;
	unsigned bit_offset = (index % 32) * 2;
	buffer[word_index] &= ~(0x3ULL << bit_offset);
	buffer[word_index] |= (uint64_t(value & 0x3) << bit_offset);
}
static void dump2bit_grid(const std::vector<uint64_t>& data, unsigned size_side, const char* title = nullptr){
	if(title) std::cout << "--- " << title << " ---\n";
	for(unsigned y = 0; y < size_side; ++y){
		for(unsigned x = 0; x < size_side; ++x){
			unsigned idx = y * size_side + x;
			unsigned val = get2bits_host(data, idx);
			if(val)
				std::cout << val << " ";
			else
				std::cout << ". ";
		}
		std::cout << "\n";
	}
	std::cout << std::endl;
}

// ---------------------------
// MAIN
// ---------------------------
int main(){
	const unsigned N = 5; // 2^5 = 32 x 32 2^12=4096
	const unsigned size_side = 1u << N;
	const unsigned total_values = size_side * size_side;
	unsigned total_words = (total_values + 31) / 32;
	const unsigned num_words = (total_values + 31) / 32;

	const size_t buffer_size = num_words * sizeof(uint64_t);
	const unsigned threads_per_block = 128;

	// Host buffers
	std::vector<uint64_t> h_input(num_words, 0);
	std::vector<uint64_t> h_result(num_words, 0);

	// Init test pattern for Test 1
	std::mt19937 rng(123);
	std::uniform_int_distribution<int> dist(0, 3);

	// --- Test 1: Set known values at edge and random positions ---
	std::cout << "[TEST 1] Boundary and random value check...\n";

	std::vector<unsigned> test_indices = {0, 1, total_values - 1};
	set2bits_host(h_input, test_indices[0], 3);
	set2bits_host(h_input, test_indices[1], 1);
	set2bits_host(h_input, test_indices[2], 2);
	dump2bit_grid(h_input, size_side, "Input");

	// Upload to device
	uint64_t* d_input, *d_output;
	CHECK_CUDA(hipMalloc(&d_input, buffer_size));
	CHECK_CUDA(hipMalloc(&d_output, buffer_size));
	CHECK_CUDA(hipMemcpy(d_input, h_input.data(), buffer_size, hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemset(d_output, 0, buffer_size));
	unsigned blocks = (total_words + threads_per_block - 1) / threads_per_block;
	int2 shift = {3, -2};
	transfer64_tile << <blocks, threads_per_block >> > (d_input, d_output, shift, size_side);
	CHECK_CUDA(hipDeviceSynchronize());
	CHECK_CUDA(hipMemcpy(h_result.data(), d_output, buffer_size, hipMemcpyDeviceToHost));
	dump2bit_grid(h_result, size_side, "Result after shift");

	// Move values back on CPU and check
	for(unsigned idx : test_indices){
		unsigned value = get2bits_host(h_input, idx);
		// compute shifted index
		unsigned x = idx % size_side;
		unsigned y = idx / size_side;
		x = (x + shift.x + size_side) % size_side;
		y = (y + shift.y + size_side) % size_side;
		unsigned shifted_idx = (y * size_side) + x;

		unsigned res_val = get2bits_host(h_result, shifted_idx);
		printf("host=%u device=%u\n", res_val, value);
		assert(res_val == value);
	}

	std::cout << "Test 1 passed ✅\n";

	// --- Test 2: Repeated shifts with sum = 0 ---
	std::cout << "[TEST 2] Shift back-and-forth + timing...\n";

	// Initialize random values
	for(unsigned i = 0; i < total_values; ++i){
		set2bits_host(h_input, i, dist(rng));
	}

	CHECK_CUDA(hipMemcpy(d_input, h_input.data(), buffer_size, hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_output, d_input, buffer_size, hipMemcpyDeviceToDevice)); // start from same

	const int2 shifts[] = {
		{5, -3}, {-2, 4}, {-3, -1}, {0, 0} // sum = 0
	};
	const int num_iters = sizeof(shifts) / sizeof(shifts[0]);

	auto start = std::chrono::high_resolution_clock::now();

	for(int i = 0; i < num_iters; ++i){
		transfer64_tile << <blocks, threads_per_block >> > 
			(d_input, d_output, shifts[i], size_side);
	}

	CHECK_CUDA(hipDeviceSynchronize());
	auto end = std::chrono::high_resolution_clock::now();

	double elapsed_ms = std::chrono::duration<double, std::milli>(end - start).count();
	std::cout << "Average time per iteration: " << (elapsed_ms / num_iters) << " ms\n";

	CHECK_CUDA(hipMemcpy(h_result.data(), d_output, buffer_size, hipMemcpyDeviceToHost));

	// Verify match
	for(unsigned i = 0; i < total_values; ++i){
		unsigned a = get2bits_host(h_input, i);
		unsigned b = get2bits_host(h_result, i);
		if(a != b){
			std::cerr << "Mismatch at index " << i << ": expected " << a << ", got " << b << "\n";
			std::exit(EXIT_FAILURE);
		}
	}

	std::cout << "Test 2 passed ✅\n";

	CHECK_CUDA(hipFree(d_input));
	CHECK_CUDA(hipFree(d_output));
	return 0;
}
